#include "hip/hip_runtime.h"
#include <iostream>
#include "raylib.h"
#include "SolverPreprocesser.hpp"
#include "Types.hpp"
#include "Window3d.hpp"
#include "Timer.hpp"
#include "Cloth.hpp"
#include "Config.hpp"

#include "D_Solver.cuh"
#include "D_Preprocessor.hpp"


int main() {

    int n;
    float size;
    float k;
    int numSubstep;
    int n_iter = 10;
    bool log_time;

    std::cout << "number of each side, cloth size, stiffness k, number of substep, log time or not" << std::endl;
    std::cin >> n >> size >> k >> numSubstep >> log_time;

    // init handles {
    hipsolverSpHandle_t cusolverSpHandle;
    hipsolverSpCreate(&cusolverSpHandle);
    hipsparseHandle_t cusparseHandle;
    hipsparseCreate(&cusparseHandle);

    // }

    // init solver {


    std::shared_ptr<Cloth> cloth = std::make_shared<Cloth>(n, size, k);

    std::shared_ptr<D_Solver> dSolver = std::make_shared<D_Solver>(cloth, n_iter);
    dSolver->SetHandles(cusolverSpHandle, cusparseHandle);

    std::shared_ptr<D_Preprocessor> pre = std::make_shared<D_Preprocessor>(dSolver);
    pre->Init();

    std::cout << ">>> Preprocessing done...\n" << std::endl;

    dSolver->AddFixed(0, 0);
    dSolver->AddFixed(0, n - 1);

    std::cout << ">>> Iteration per substep: " << n_iter << std::endl << std::endl;

    // test 3
//    std::cout << "row ptr:" << std::endl;
//    for (int i = 0; i < dSolver->h_M.outerSize() + 1; i++) {
//        std::cout << dSolver->d_M.d_row_ptr_vec[i] << std::endl;
//    }
//    std::cout << "\n\ncol idx:" << std::endl;
//    for (int i = 0; i < dSolver->h_M.nonZeros(); i++) {
//        std::cout << dSolver->d_M.d_col_idx_vec[i] << std::endl;
//    }
//    std::cout << "\n\nval:" << std::endl;
//    for (int i = 0; i < dSolver->h_M.nonZeros(); i++) {
//        std::cout << dSolver->d_M.d_val_vec[i] << std::endl;
//    }



    // test 2
//    std::cout << "row ptr:" << std::endl;
//    for (int i = 0; i < dSolver->h_M.outerSize() + 1; i++) {
//        std::cout << dSolver->h_M.outerIndexPtr()[i] << std::endl;
//    }
//
//    std::cout << "\n\ncol idx:" << std::endl;
//    for (int i = 0; i < dSolver->h_M.nonZeros(); i++) {
//        std::cout << dSolver->h_M.innerIndexPtr()[i] << std::endl;
//    }
//
//    std::cout << "\n\nval:" << std::endl;
//    for (int i = 0; i < dSolver->h_M.nonZeros(); i++) {
//        std::cout << dSolver->h_M.valuePtr()[i] << std::endl;
//    }


    bool GUI;
    std::cout << "enable GUI?" << std::endl;
    std::cin >> GUI;


    // }

//     def update {

    auto update = [&] {

        for (int substep = 0; substep < numSubstep; substep++) {

//            std::cout << "substep :" << substep << std::endl;
            dSolver->Step();

        }
    };

    // }

    Timer updateTimer, drawTimer;

    if (GUI) {
//        const int screenWidth = 1200;
//        const int screenHeight = 1200;
//        const float radius = size / static_cast<float>(n) * .15f;
//
//        InitWindow(screenWidth, screenHeight, "Projective dynamimcs");
//        SetTargetFPS(60);
//        Window3d window3d;
//        window3d.Init();
//        while (not WindowShouldClose()) {
//
//
//            updateTimer.start();
//            if (!window3d.pause) {
//                update();
//
//            }
//            updateTimer.stop();
////        std::cout << ">>> update" << std::endl;
//
//
//            BeginDrawing();
//            ClearBackground(RAYWHITE);
//
//            window3d.Update();
//            window3d.Begin();
//
//            drawTimer.start();
//            for (int i = 0; i < hSolver->cloth->numVertex; i++) {
//
//                Vector3 center = {hSolver->x[3 * i], hSolver->x[3 * i + 1], hSolver->x[3 * i + 2]};
//                DrawPoint3D(center, RED);
////            DrawSphere(center, radius, RED);
//
//            }
//            drawTimer.stop();
//
//
//            window3d.End();
//
//
//            EndDrawing();
//
//            if (log_time) {
//                printf("update time: %f, draw time: %f\n\n", updateTimer.getTime(), drawTimer.getTime());
//            }
//
//
//        }
//
//        CloseWindow();
    } else {
        while (true) {


            updateTimer.start();
            update();
            updateTimer.stop();
            std::cout << ">>> update" << std::endl;


            for (int irow = 0; irow < dSolver->cloth->nside; irow++) {
                for (int icol = 0; icol < dSolver->cloth->nside; icol++) {
                    printf("[%.3f, %.3f, %.3f]  ",
                           dSolver->h_x[3 * dSolver->index(irow, icol)],
                           dSolver->h_x[3 * dSolver->index(irow, icol) + 1],
                           dSolver->h_x[3 * dSolver->index(irow, icol) + 2]);
                }
                std::cout << std::endl;
            }

            std::cout << "\n --- Press any key to continue --- \n";
            std::cin.get();


        }
    }


    return 0;

}


